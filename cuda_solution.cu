// #include <stdio.h>
// #include <stdlib.h>
// #include <time.h>
// #include <sys/time.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>


// 32x32 Threads in a block.
#define NTHREADS_X 32
#define NTHREADS_Y 32
#define THREADS_PER_BLOCK NTHREADS_X * NTHREADS_Y

/* 
* A macro used for error checking in CUDA function calls
* http://stackoverflow.com/a/14038590
*/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void matrix_mul(int *a, int *b, int *c, int a_ncolumns, int c_nlines,
        int c_ncolumns, int nBlocks) {
    int i, z, cur_value = 0;

    // number of multiplications
    int nMultiplications = a_ncolumns;

    // multiplications per block
    int multiplicationsInBlock = NTHREADS_Y;

    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int line =  blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ int s_a[NTHREADS_Y][NTHREADS_X];
    __shared__ int s_b[NTHREADS_Y][NTHREADS_X];

    int a_tLine, a_tColumn, b_tLine, b_tColumn;

    for (z = 0; z < nBlocks; z++) {

        // Load Matrix A
        a_tLine = (blockIdx.y * NTHREADS_Y + threadIdx.y);
        a_tColumn = (z * NTHREADS_X + threadIdx.x);
        if (a_tLine < c_nlines && a_tColumn < a_ncolumns) {
            s_a[threadIdx.y][threadIdx.x] = a[ (a_ncolumns * a_tLine) + a_tColumn];
        }

        // Load Matrix B
        b_tLine = (z * NTHREADS_Y + threadIdx.y);
        b_tColumn = (blockIdx.x * NTHREADS_X + threadIdx.x);
        if (b_tLine < a_ncolumns && b_tColumn < c_ncolumns) {
            s_b[threadIdx.y][threadIdx.x] = b[ (c_ncolumns * b_tLine) + b_tColumn ];
        }

        __syncthreads();

	// Checkin position in Matrix C
        if (column < c_ncolumns && line < c_nlines) {
            if (nMultiplications < NTHREADS_Y) {
                multiplicationsInBlock = nMultiplications;
            }

            for (i = 0; i < multiplicationsInBlock; i++) {
                cur_value |= s_a[threadIdx.y][i] && s_b[i][threadIdx.x];
            }

            nMultiplications -= NTHREADS_Y;
        }

        __syncthreads();
    }

    // Set value to Matrix C
    bool has_changed = false;
    if (column < c_ncolumns && line < c_nlines) {
	has_changed |= (cur_value != c[line * c_ncolumns + column]);
        c[line * c_ncolumns + column] |= cur_value;
    }
    // return has_changed;
}

int main()
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int a_nlines, a_ncolumns;
    int b_nlines, b_ncolumns;
    int c_nlines, c_ncolumns;

    size_t a_size, b_size, c_size;
    int i, j;

    hipEvent_t start, stop;
    gpuErrchk( hipEventCreate(&start) );
    gpuErrchk( hipEventCreate(&stop) );

//    scanf("%d", &a_nlines);
//    scanf("%d", &a_ncolumns);
//    scanf("%d", &b_nlines);
//    scanf("%d", &b_ncolumns);
    std::cin >> a_nlines >> a_ncolumns >> b_nlines >> b_ncolumns;

    c_nlines = a_nlines;
    c_ncolumns = b_ncolumns;

#ifdef __DEBUG
    printf("a_nlines: %d\na_ncolumns: %d\nb_nlines: %d\nb_ncolumns: %d\nc_nlines: %d\nc_ncolumns: %d\n", a_nlines, a_ncolumns, b_nlines, b_ncolumns, c_nlines, c_ncolumns);
#endif

    if ( a_ncolumns != b_nlines ) {
	std::cout << "Number of columns in Matrix A should be equals to number of lines in Matrix B\n";
        return EXIT_FAILURE;
    }

    a_size = a_nlines * a_ncolumns * sizeof(int);
    b_size = b_nlines * b_ncolumns * sizeof(int);
    c_size = c_nlines * c_ncolumns * sizeof(int);

    gpuErrchk( hipMalloc((void **) &d_a, a_size) );
    gpuErrchk( hipMalloc((void **) &d_b, b_size) );
    gpuErrchk( hipMalloc((void **) &d_c, c_size) );

    a = (int *)malloc(a_size);
    b = (int *)malloc(b_size);
    c = (int *)malloc(c_size);

    memset(c, 0, c_nlines*c_ncolumns*sizeof(int));

    for (i = 0; i < a_nlines; i++)
    {
        for (j = 0; j < a_ncolumns; j++)
        {
            scanf("%d", &a[i * a_ncolumns + j]);
        }
    }

    for (i = 0; i < b_nlines; i++)
    {
        for (j = 0; j < b_ncolumns; j++)
        {
            scanf("%d", &b[i * b_ncolumns + j]);
        }
    }

    gpuErrchk( hipMemcpy(d_a, a, a_size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_b, b, b_size, hipMemcpyHostToDevice) );

    dim3 tbloco = dim3(
                    (int) std::ceil( (double) c_ncolumns / NTHREADS_X ),
                    (int) std::ceil( (double) c_nlines / NTHREADS_Y ),
                    1
                );

    dim3 tthreads = dim3(
                        NTHREADS_X,
                        NTHREADS_Y,
                        1
                    );

#ifdef __DEBUG
    printf("tbloco.x: %d tbloco.y: %d tbloco.z: %d\n", tbloco.x, tbloco.y, tbloco.z);
    printf("tthreads.x: %d tthreads.y: %d\n", tthreads.x, tthreads.y);
#endif

    hipEventRecord(start);

    // kernel call
    matrix_mul<<<tbloco,tthreads>>>(d_a, d_b, d_c, a_ncolumns, c_nlines,
        c_ncolumns, (int) std::ceil( (double) a_ncolumns / NTHREADS_X));

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipEventRecord(stop) );
    gpuErrchk( hipMemcpy(c, d_c, c_size, hipMemcpyDeviceToHost) );
    gpuErrchk( hipEventSynchronize(stop) );

#ifndef __NO_OUTPUT
    // print Matrix
    for (i = 0; i < c_nlines; i++)
    {
        for (j = 0; j < c_ncolumns; j++)
        {
            printf("%d ", c[i * c_ncolumns + j]);
        }
        printf("\n");
    }
    printf("\n");
#endif

#ifdef __TIME
    float milliseconds = 0;
    gpuErrchk( cudaEventElapsedTime(&milliseconds, start, stop) );
    printf("%.5f\n", milliseconds);
#endif

    free(a); free(b); free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
